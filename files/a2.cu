#include <hip/hip_runtime.h>
#include <stdio.h>

#define QWA 1000

__global__ void compress_delta_cuda(const int *input, int *output, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < n; i += stride) {
        int val = input[i];
        output[i] = val - QWA;
    }
}

// Host launcher (with full error checks and timing for profiling)
void run_cuda(const int *h_input, int *h_output, int n) {
    int *d_input, *d_output;

    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));

    hipMemcpy(d_input, h_input, n * sizeof(int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    compress_delta_cuda<<<blocks, threads>>>(d_input, d_output, n);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
        return;
    }

    hipMemcpy(h_output, d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

